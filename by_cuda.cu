
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

#define BLOCK_SIZE 10
uint32_t power_of_2(uint32_t val)
{
	if ((val & (val - 1)) == 0) {
		return val;
	}
	uint32_t andv = 0x80000000;

	while ((andv & val) == 0) {
		andv >>= 1;
	}

	return andv << 1;
}

__device__ uint32_t power_of_2_d(uint32_t val)
{
	if ((val & (val - 1)) == 0) {
		return val;
	}
	uint32_t andv = 0x80000000;

	while ((andv & val) == 0) {
		andv >>= 1;
	}

	return andv << 1;
}

__global__ void work_efficient_scan_kernel(int32_t* X, ssize_t InputSize,
                                           int32_t* Y)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	for (ssize_t stride = 2; stride <= InputSize; stride *= 2) {
		__syncthreads();
		if (i > 0 && i % stride == (stride - 1) && i < InputSize) {
			X[i] += X[i - stride / 2];
		}
	}

	for (ssize_t stride = InputSize; stride >= 1; stride /= 2) {
		__syncthreads();

		if (i < InputSize && ((i + 1) % stride) == 0 &&
		    ((i + 1) % (stride * 2)) != 0 && power_of_2_d(i + 1) != i + 1) {

			int32_t sum = 0;
			ssize_t pos = 0;

			for (int x = InputSize; x > 0; x >>= 1) {
				if ((x & (i + 1)) != 0) {
					sum += X[x + pos - 1];
					pos += x;
				}
			}

			X[i] = sum;
		}
	}

	__syncthreads();
	if (i < InputSize) {
		Y[i] = X[threadIdx.x];
	}
}

void prefix_by_cuda(int32_t* input, ssize_t n, int32_t* output)
{

	int32_t *indev, *outdev;
	uint32_t power = power_of_2(n);

	printf("power=%u\n", power);
	hipMalloc(&indev, sizeof(int32_t) * power);
	hipMalloc(&outdev, sizeof(int32_t) * n);

	hipMemcpy(indev, input, sizeof(int32_t) * n, hipMemcpyHostToDevice);
	hipMemset(indev + n, 0, power - n);

	work_efficient_scan_kernel<<<64, 64>>>(indev, power, outdev);

	hipMemcpy(output, outdev, sizeof(int32_t) * n, hipMemcpyDeviceToHost);

	hipFree(indev);
	hipFree(outdev);
}