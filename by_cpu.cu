
#include <hip/hip_runtime.h>
void prefix_by_cpu(int32_t* input, ssize_t n, int32_t* output)
{
	int32_t sum = 0;

	for (ssize_t i = 0; i < n; i++) {
		sum += input[i];
		output[i] = sum;
	}
}
